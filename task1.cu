#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#define CEIL(a, b) ((a-1)/b +1)
const int ROWS = 30;
const int COLUMNS = 30;
const int BLOCK_SIZE = 1024;
const int MATRIX_BYTES = ROWS*COLUMNS*sizeof(int);

__global__ void get_matrix(int *d_in, int *d_out){
  
  int idx = threadIdx.x;
  
  int m = ROWS>COLUMNS?COLUMNS:ROWS;
  int n = ROWS>COLUMNS?ROWS:COLUMNS;
  m--;n--;
  int n_iter = m+n-1;
  int tbc,left,up,corner;

  for(int i=0;i<n_iter;i++){
    int x_index,y_index;
    if(i >= n){
      // FOR THIRD PART OF ITERATIONS
      
      if(idx >= i-n+1){
        if(ROWS<=COLUMNS){
            x_index = i+1-idx+m-n;
            y_index = idx+1+n-m;
        }
        else{
            x_index = idx+1+n-m;
            y_index = i+1-idx+m-n;
        }
        tbc = *(d_in + ((x_index) * COLUMNS + (y_index)));
        if(!tbc){
          *(d_out + ((x_index) * COLUMNS + (y_index))) = 0;
        }
        else{
          left = *(d_out + ((x_index) * COLUMNS + (y_index-1)));
          up = *(d_out + ((x_index-1) * COLUMNS + (y_index)));
          corner = *(d_out + ((x_index-1) * COLUMNS + (y_index-1)));
          int mini = (left>up?up:left);
          *(d_out + ((x_index) * COLUMNS + (y_index))) = (mini>corner?corner:mini)+1;
        }
      }
    }
    else if(i>=m-1 && i<=n-1){
      // FOR SECOND PART OF ITERATIONS

      if(ROWS<=COLUMNS){
          x_index = m-idx;
          y_index = idx+i-m+2;
      }
      else{
          x_index = idx+i-m+2;
          y_index = m-idx;
      }
        tbc = *(d_in + ((x_index) * COLUMNS + (y_index)));
          if(!tbc)
            *(d_out + ((x_index) * COLUMNS + (y_index))) = 0;
          else{
            left = *(d_out + ((x_index) * COLUMNS + (y_index-1)));
            up = *(d_out + ((x_index-1) * COLUMNS + (y_index)));
            corner = *(d_out + ((x_index-1) * COLUMNS + (y_index-1)));
            int mini = (left>up?up:left);
            *(d_out + ((x_index) * COLUMNS + (y_index))) = (mini>corner?corner:mini)+1;
          }
    }
    else{
      // FOR FIRST PART OF ITERATIONS
      if(idx <= i){
        tbc = *(d_in + ((i+1-idx) * COLUMNS + (idx+1)));
        if(!tbc)
          *(d_out + ((i+1-idx) * COLUMNS + (idx+1))) = 0;
        else{
          left = *(d_out + ((i+1-idx) * COLUMNS + (idx)));
          up = *(d_out + ((i-idx) * COLUMNS + (idx+1)));
          corner = *(d_out + ((i-idx) * COLUMNS + (idx)));
          int mini = (left>up?up:left);
          *(d_out + ((i+1-idx) * COLUMNS + (idx+1))) = (mini>corner?corner:mini)+1;
        }
      }
    }
    __syncthreads();
  }

}

int S[ROWS][COLUMNS];

struct combine{
    int max;
    int *ptr;
};
struct combine ans_cpu;

void cpu_method(int *h_in_element){
    int i,j;      
    for(i = 0; i < ROWS; i++)  
        S[i][0] = *(h_in_element + i*COLUMNS);  
      
    for(j = 0; j < COLUMNS; j++)  
        S[0][j] = *(h_in_element + j); 
          
    for(i = 1; i < ROWS; i++){  
        for(j = 1; j < COLUMNS; j++){  
            if(*(h_in_element + (i*COLUMNS+j)) == 1){
                int left = S[i][j-1];
                int up = S[i-1][j];
                int corner = S[i-1][j-1];
                int mini = (left>up?up:left);  
                S[i][j] = (mini>corner?corner:mini) + 1;
              }  
            else
                S[i][j] = 0;  
        }  
    }
    int max_of_s = S[0][0], max_i, max_j;  
    for(i = 0; i < ROWS; i++){  
        for(j = 0; j < COLUMNS; j++){  
            if(max_of_s < S[i][j]){
                max_of_s = S[i][j];  
                max_i = i;  
                max_j = j;  
            }  
        }              
    }
    ans_cpu.max = max_of_s;
    ans_cpu.ptr = &S[0][0];  
}

int test_solution(int *ptr, int *h_out, int *h_in, int gpu_result, int cpu_result){
    int flag = 1;
    for(int i=0;i<ROWS;i++){
      int j;
      for(j=0;j<COLUMNS;j++){
        if(*(ptr + (i*COLUMNS+j)) != *(h_out + (i*COLUMNS+j))){
            //printf("%d, %d\n",i,j);
            flag = 0;
            break;
        }
      }
      if(j!=COLUMNS)
          break;
    }
    /*for(int i=0;i<ROWS;i++){
      for(int j=0;j<COLUMNS;j++){
        printf("%d ",*(h_in + (i*COLUMNS+j)));
      }
      printf("\n");
    }

    printf("\n");

    for(int i=0;i<ROWS;i++){
      for(int j=0;j<COLUMNS;j++){
        printf("%d ",*(ptr + (i*COLUMNS+j)));
      }
      printf("\n");
    }

    printf("\n");

    for(int i=0;i<ROWS;i++){
      for(int j=0;j<COLUMNS;j++){
        printf("%d ",*(h_out + (i*COLUMNS+j)));
      }
      printf("\n");
    }*/

    if(gpu_result != cpu_result){
        flag = 0;
    }
    return flag;
}

__global__ void find_max(int *d_final, int *d_max){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < ROWS*COLUMNS){
        int step = 1;
        while(step < blockDim.x){
            if(idx % (step*2) == 0 && (idx+step)<(ROWS*COLUMNS)){
                *(d_final + idx) = *(d_final + idx) > *(d_final + idx + step) ? *(d_final + idx) : *(d_final + idx + step);
            }
            __syncthreads();
            step *= 2;
        }
        if(threadIdx.x==0){
            atomicMax(d_max, *(d_final + idx));
        }
    }
}

int main(){

  srand(time(0));

  int h_in[ROWS*COLUMNS], h_out[ROWS*COLUMNS];

  for(int i=0;i<ROWS;i++){
    for(int j=0;j<COLUMNS;j++){
          *(h_in + (i*COLUMNS+j)) = rand() % 2;
    }
  }

  for(int i=0;i<COLUMNS;i++)
    *(h_out + (i)) = *(h_in + (i));

  for(int i=0;i<ROWS;i++)
    *(h_out + (i*COLUMNS)) = *(h_in + (i*COLUMNS));

  for(int i=1;i<ROWS;i++){
    for(int j=1;j<COLUMNS;j++){
      *(h_out + (i*COLUMNS+j)) = -1;
    }
  }

  int *d_in, *d_out, *d_final, *d_max;

  hipMalloc(&d_in, MATRIX_BYTES);
  hipMalloc(&d_out, MATRIX_BYTES);
  hipMalloc(&d_final, MATRIX_BYTES);
  hipMalloc(&d_max, sizeof(int));

  hipMemcpy(d_in, h_in, MATRIX_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_out, h_out, MATRIX_BYTES, hipMemcpyHostToDevice);

  int min = ROWS>=COLUMNS?COLUMNS:ROWS;
  int THREADS;
  if(min>=1025)
    THREADS = 1024;
  else
    THREADS = min - 1;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int *h_max;
  hipEventRecord(start);

  get_matrix<<<1,THREADS>>>(d_in, d_out);
  hipMemcpy(h_out, d_out, MATRIX_BYTES, hipMemcpyDeviceToHost);
  hipMemcpy(d_final, h_out, MATRIX_BYTES, hipMemcpyHostToDevice);
  find_max<<<CEIL((ROWS*COLUMNS), BLOCK_SIZE), BLOCK_SIZE>>>(d_final, d_max);

  hipEventRecord(stop);

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  hipMemcpy(h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
  int gpu_result = *h_max;

  clock_t cpu_startTime, cpu_endTime;
  double cpu_ElapseTime=0;
  cpu_startTime = clock();

  cpu_method(&h_in[0]);
  int *ptr = ans_cpu.ptr;
  int cpu_result = ans_cpu.max;

  cpu_endTime = clock();
  cpu_ElapseTime = ((cpu_endTime - cpu_startTime)/(1.0 * CLOCKS_PER_SEC)) * 1000;

  int flag = test_solution(ptr, &h_out[0], &h_in[0], gpu_result, cpu_result);

  if(flag) {
    printf("The computed matrix is correct!\n");
    printf("Time taken by GPU : %f ms\n", milliseconds);
    printf("Time taken by CPU : %f ms\n", cpu_ElapseTime);
    printf("The area of square is %d sq units\n",gpu_result*gpu_result);
  }
  else {
    printf("The computed matrix is incorrect!\n");
  }

  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_final);
  hipFree(d_max);
  return 0;

}
